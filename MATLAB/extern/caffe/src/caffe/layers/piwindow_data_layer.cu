// Copyright 2014 BVLC and contributors.
//
// Based on data_layer.cpp by Yangqing Jia.

#include <stdint.h>
#include <pthread.h>

#include <string>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/vision_layers.hpp"

using std::string;
using std::map;
using std::pair;
namespace caffe {

template <typename Dtype>
Dtype PiWindowDataLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  // First, join the thread
  JoinPrefetchThread();
  // Copy the data
  CUDA_CHECK(hipMemcpy((*top)[0]->mutable_gpu_data(),
      prefetch_box_->cpu_data(), sizeof(Dtype) * prefetch_box_->count(),
      hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy((*top)[1]->mutable_gpu_data(),
      prefetch_reg_->cpu_data(), sizeof(Dtype) * prefetch_reg_->count(),
      hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy((*top)[2]->mutable_gpu_data(),
      prefetch_label_->cpu_data(), sizeof(Dtype) * prefetch_label_->count(),
      hipMemcpyHostToDevice));
  // Start a new prefetch thread
  CreatePrefetchThread();
  return Dtype(0.);  
}

INSTANTIATE_CLASS(PiWindowDataLayer);

}
